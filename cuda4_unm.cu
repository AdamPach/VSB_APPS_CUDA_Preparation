#include "hip/hip_runtime.h"
// ***********************************************************************
//
// Demo program for education in subject
// Computer Architectures and Parallel Systems.
// Petr Olivka, dep. of Computer Science, FEI, VSB-TU Ostrava, 2020/11
// email:petr.olivka@vsb.cz
//
// Example of CUDA Technology Usage with unified memory.
//
// Image transformation from RGB to BW schema. 
//
// ***********************************************************************

#include <stdio.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>

#include "cuda_img.h"

// Demo kernel to transform RGB color schema to BW schema
__global__ void kernel_grayscale( CudaImg t_color_cuda_img, CudaImg t_bw_cuda_img )
{
    // X,Y coordinates and check image dimensions
    int l_y = blockDim.y * blockIdx.y + threadIdx.y;
    int l_x = blockDim.x * blockIdx.x + threadIdx.x;
    if ( l_y >= t_color_cuda_img.m_size.y ) return;
    if ( l_x >= t_color_cuda_img.m_size.x ) return;

    // Get point from color picture
    uchar3 l_bgr = t_color_cuda_img.m_p_uchar3[ l_y * t_color_cuda_img.m_size.x + l_x ];

    // Store BW point to new image
   // t_bw_cuda_img.m_p_uchar1[ l_y * t_bw_cuda_img.m_size.x + l_x ].x = l_bgr.x * 0.11 + l_bgr.y * 0.59 + l_bgr.z * 0.30;
   int flip_index = t_color_cuda_img.m_size.x - l_x;
   t_bw_cuda_img.m_p_uchar3[ l_y * t_bw_cuda_img.m_size.x + flip_index ] = l_bgr ;
}

void cu_run_grayscale( CudaImg t_color_cuda_img, CudaImg t_bw_cuda_img )
{
    hipError_t l_cerr;

    // Grid creation, size of grid must be equal or greater than images
    int l_block_size = 16;
    dim3 l_blocks( ( t_color_cuda_img.m_size.x + l_block_size - 1 ) / l_block_size, ( t_color_cuda_img.m_size.y + l_block_size - 1 ) / l_block_size );
    dim3 l_threads( l_block_size, l_block_size );
    kernel_grayscale<<< l_blocks, l_threads >>>( t_color_cuda_img, t_bw_cuda_img );

    if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

    hipDeviceSynchronize();
}

//rotation 90 clockwise


__global__ void kernel_rotate_90_clockwise(CudaImg input_cuda_img, CudaImg output_cuda_img)
{
    int l_y = blockDim.y * blockIdx.y + threadIdx.y;
    int l_x = blockDim.x * blockIdx.x + threadIdx.x;
    if ( l_y >= input_cuda_img.m_size.y ) return;
    if ( l_x >= input_cuda_img.m_size.x ) return;
   
    int new_x = input_cuda_img.m_size.y - l_y;
    int new_y = l_x;

    output_cuda_img.at4(new_x, new_y) = input_cuda_img.at4(l_x, l_y);
}

void cu_rotate_90_clockwise(CudaImg input_cuda_img, CudaImg output_cuda_img)
{
    hipError_t cuda_error;

    int l_block_size = 16;
    dim3 l_blocks( ( input_cuda_img.m_size.x + l_block_size - 1 ) / l_block_size, ( input_cuda_img.m_size.y + l_block_size - 1 ) / l_block_size );
    dim3 l_threads( l_block_size, l_block_size );

    kernel_rotate_90_clockwise<<<l_blocks, l_threads>>>(input_cuda_img, output_cuda_img);

    if ( ( cuda_error = hipGetLastError() ) != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cuda_error ) );

    hipDeviceSynchronize();
}

__global__ void kernel_rotate_90_anticlockwise(CudaImg input_cuda_img, CudaImg output_cuda_img)
{
    int l_y = blockDim.y * blockIdx.y + threadIdx.y;
    int l_x = blockDim.x * blockIdx.x + threadIdx.x;
    if ( l_y >= input_cuda_img.m_size.y ) return;
    if ( l_x >= input_cuda_img.m_size.x ) return;
   
    int new_x = l_y;
    int new_y = input_cuda_img.m_size.x - l_x;

    output_cuda_img.at4(new_x, new_y) = input_cuda_img.at4(l_x, l_y);
}

void cu_rotate_90_anticlokwise(CudaImg input_cuda_img, CudaImg output_cuda_img)
{
    hipError_t cuda_error;

    int l_block_size = 16;
    dim3 l_blocks( ( input_cuda_img.m_size.x + l_block_size - 1 ) / l_block_size, ( input_cuda_img.m_size.y + l_block_size - 1 ) / l_block_size );
    dim3 l_threads( l_block_size, l_block_size );

    kernel_rotate_90_anticlockwise<<<l_blocks, l_threads>>>(input_cuda_img, output_cuda_img);

    if ( ( cuda_error = hipGetLastError() ) != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cuda_error ) );

    hipDeviceSynchronize();
}
